#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <png++/png.hpp>

int THREAD_PER_DIM = 32;

struct
{
    size_t width;
    size_t height;
    unsigned char *data;
} typedef Image;

/**
 * @brief Reads an image from a file and returns an Image object.
 *
 * @param filename The path to the image file.
 * @return Image* A pointer to the Image object containing the image data.
 */
Image *read_image(const char *filename)
{
    png::image<png::rgb_pixel> image(filename);

    size_t width = image.get_width();
    size_t height = image.get_height();

    auto pixbuf = image.get_pixbuf();

    unsigned char *data = (unsigned char *)malloc(width * height * 3);

    for (size_t row = 0; row < height; row++)
    {
        for (size_t col = 0; col < width; col++)
        {
            auto pixel = pixbuf[row][col];
            data[(row * width + col) * 3] = pixel.red;
            data[(row * width + col) * 3 + 1] = pixel.green;
            data[(row * width + col) * 3 + 2] = pixel.blue;
        }
    }

    Image *img = (Image *)malloc(sizeof(Image));
    img->width = width;
    img->height = height;
    img->data = data;

    return img;
}

/**
 * @brief Initializes a grayscale image with the specified width and height.
 *
 * @param width The width of the image.
 * @param height The height of the image.
 * @return A pointer to the initialized Image structure.
 */
Image *initialize_grayscale_image(size_t width, size_t height)
{
    Image *img = (Image *)malloc(sizeof(Image));
    img->width = width;
    img->height = height;
    img->data = (unsigned char *)malloc(width * height);

    for (size_t i = 0; i < width * height; i++)
    {
        img->data[i] = 0;
    }

    return img;
}

/**
 * Prints the RGB values and corresponding gray value of each pixel in the given images.
 *
 * @param colour The input image containing RGB values.
 * @param gray The output image containing gray values.
 */
void print_image(const Image *colour, const Image *gray)
{
    for (size_t i = 0; i < colour->width * colour->height; i++)
    {
        printf("R: %u, G: %u, B: %u, Gray Value: %u\n", colour->data[i * 3], colour->data[i * 3 + 1], colour->data[i * 3 + 2], gray->data[i]);
    }
}

__global__ void grayscale(const unsigned char *input, unsigned char *output, size_t width, size_t height)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height)
    {
        int index = row * width + col;
        unsigned char r = input[index * 3];
        unsigned char g = input[index * 3 + 1];
        unsigned char b = input[index * 3 + 2];

        output[index] = 0.21f * r + 0.71f * g + 0.07f * b;
    }
}

void grayscale_cpu(const unsigned char *input, unsigned char *output, size_t width, size_t height)
{
    for (size_t row = 0; row < height; row++)
    {
        for (size_t col = 0; col < width; col++)
        {
            int index = row * width + col;
            unsigned char r = input[index * 3];
            unsigned char g = input[index * 3 + 1];
            unsigned char b = input[index * 3 + 2];

            output[index] = 0.21f * r + 0.71f * g + 0.07f * b;
        }
    }
}

int main(int argc, char **argv)
{
    bool cuda = false;
    if (argc > 1)
    {
        std::string arg1 = argv[1];
        if (arg1 == "cuda")
        {
            cuda = true;
        }
        else
        {
            cuda = false;
        }
    }
    Image *lenna_h = read_image("lenna.png");
    Image *gray_h = initialize_grayscale_image(lenna_h->width, lenna_h->height);

    if (cuda)
    {
        printf("Running on GPU\n");
        unsigned char *lenna_data_d;
        unsigned char *gray_data_d;
        hipError_t error;

        error = hipMalloc((void **)&lenna_data_d, lenna_h->width * lenna_h->height * 3);
        if (error != hipSuccess)
        {
            printf("@ line %d : error allocating memory for lenna data: %s\n", __LINE__, hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }

        error = hipMalloc((void **)&gray_data_d, lenna_h->width * lenna_h->height);
        if (error != hipSuccess)
        {
            printf("@ line %d : error allocating memory for gray data: %s\n", __LINE__, hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }

        hipMemcpy(lenna_data_d, lenna_h->data, lenna_h->width * lenna_h->height * 3, hipMemcpyHostToDevice);
        hipMemcpy(gray_data_d, gray_h->data, lenna_h->width * lenna_h->height, hipMemcpyHostToDevice);

        dim3 blocksPerGrid(ceil(lenna_h->width) / THREAD_PER_DIM, ceil(lenna_h->height) / THREAD_PER_DIM);
        dim3 threadsPerBlock(THREAD_PER_DIM, THREAD_PER_DIM);

        grayscale<<<blocksPerGrid, threadsPerBlock>>>(lenna_data_d, gray_data_d, lenna_h->width, lenna_h->height);

        hipMemcpy(gray_h->data, gray_data_d, lenna_h->width * lenna_h->height, hipMemcpyDeviceToHost);
        hipFree(lenna_data_d);
        hipFree(gray_data_d);

    }
    else
    {
        printf("Running on CPU\n");
        grayscale_cpu(lenna_h->data, gray_h->data, lenna_h->width, lenna_h->height);
    }
    return 0;
}